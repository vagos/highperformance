#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

#ifndef PROBDIM
#define PROBDIM 2
#endif

#include "func.h"

static double **xdata;
static double ydata[TRAINELEMS];

hipError_t err = hipSuccess;

#define MAX_NNB	256

__global__ void compute_dist(double *xdata, double *q, int npat, int lpat, double *dist)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < npat) {
        double sum = 0.0;
        for (int j = 0; j < lpat; j++) {
            double diff = xdata[i*lpat+j] - q[j];
            sum += diff*diff;
        }
        dist[i] = sum;
    }
}

void compute_knn_brute_force(double *xdata, double *q,double *d_dist, int npat, int lpat, int knn, int *nn_x, double *nn_d, double *dist)
{
	int i, max_i;
	double max_d, new_d;


	// initialize pairs of index and distance 
	for (i = 0; i < knn; i++) {
		nn_x[i] = -1;
		nn_d[i] = 1e99-i;
	}

	// max_d = compute_max_pos(nn_d, knn, &max_i);

    // create array of npat elements with hipHostAlloc
	// err = hipHostAlloc((void **)&dist, npat*sizeof(double), hipHostMallocDefault);
	// if(err != hipSuccess) {
	// 	fprintf(stderr, "Failed to allocate host vector dist (error code %s)! %d\n", hipGetErrorString(err), __LINE__);
	// 	exit(EXIT_FAILURE);
	// }

    int threadsPerBlock = 256;
    int blocksPerGrid = (npat + threadsPerBlock - 1) / threadsPerBlock;

    // compute distances to all points
    compute_dist<<<blocksPerGrid, threadsPerBlock>>>(xdata, q, npat, lpat, d_dist);

    hipDeviceSynchronize();
    hipMemcpy(dist, d_dist, npat*sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if(err != hipSuccess) {
        fprintf(stderr, "Failed to launch compute_dist kernel (error code %s)! %d", hipGetErrorString(err), __LINE__);
        exit(EXIT_FAILURE);
    }

    // compute distances to all points
    // for (i = 0; i < npat; i++) {
    //     dist[i] = compute_dist(q, xdata[i], lpat);
    // }

	for (i = 0; i < npat; i++) {
		new_d = dist[i];	// euclidean
		if (new_d < max_d) {	// add point to the  list of knns, replace element max_i
			max_d = compute_max_pos(nn_d, knn, &max_i);
			nn_x[max_i] = i;
			nn_d[max_i] = new_d;
		}
		
	}

	// sort the knn list 

    int j;
	int temp_x;
	double temp_d;

	for (i = (knn - 1); i > 0; i--) {
		for (j = 1; j <= i; j++) {
			if (nn_d[j-1] > nn_d[j]) {
				temp_d = nn_d[j-1]; nn_d[j-1] = nn_d[j]; nn_d[j] = temp_d;
				temp_x = nn_x[j-1]; nn_x[j-1] = nn_x[j]; nn_x[j] = temp_x;
			}
		}
	}

	return;
}


/* compute an approximation based on the values of the neighbors */
double predict_value(int dim, int knn, double *xdata, double *ydata, double *point, double *dist)
{
	int i;
	double sum_v = 0.0;
	// plain mean (other possible options: inverse distance weight, closest value inheritance)

	for (i = 0; i < knn; i++) {
		sum_v += ydata[i];
	}

	return sum_v/knn;
}


double find_knn_value(double *d_xmem, double *d_x, double *d_dist,double *p, int n, int knn, double *dist)
{
	int nn_x[MAX_NNB];
	double nn_d[MAX_NNB];

	compute_knn_brute_force(d_xmem, d_x, d_dist, TRAINELEMS, PROBDIM, knn, nn_x, nn_d, dist); // brute-force /linear search

	int dim = PROBDIM;
	int nd = knn;   // number of points
	double xd[MAX_NNB*PROBDIM];   // points
	double fd[MAX_NNB];     // function values

	for (int i = 0; i < knn; i++) {
		fd[i] = ydata[nn_x[i]];
	}

	for (int i = 0; i < knn; i++) {
		for (int j = 0; j < PROBDIM; j++) {
			xd[i*dim+j] = xdata[nn_x[i]][j];
		}
	}

	double fi;

	fi = predict_value(dim, nd, xd, fd, p, nn_d);

	return fi;
}

int main(int argc, char *argv[])
{
	if (argc != 3)
	{
		printf("usage: %s <trainfile> <queryfile>\n", argv[0]);
		exit(1);
	}

	char *trainfile = argv[1];
	char *queryfile = argv[2];

	double *xmem = (double *)malloc(TRAINELEMS*PROBDIM*sizeof(double));
	xdata = (double **)malloc(TRAINELEMS*sizeof(double *));
	for (int i = 0; i < TRAINELEMS; i++) xdata[i] = xmem + i*PROBDIM; //&xmem[i*PROBDIM];

	FILE *fpin = open_traindata(trainfile);

	for (int i=0;i<TRAINELEMS;i++) {
		for (int k = 0; k < PROBDIM; k++)
            xdata[i][k] = read_nextnum(fpin);

#if defined(SURROGATES)
        ydata[i] = read_nextnum(fpin);
#else
        ydata[i] = 0;
#endif
	}
	fclose(fpin);

    /* Read query data */

	fpin = open_querydata(queryfile);

	double *y = (double *)malloc(QUERYELEMS*sizeof(double));
	double *x = (double *)malloc(QUERYELEMS*PROBDIM*sizeof(double));

	for (int i=0;i<QUERYELEMS;i++) {	/* requests */

        for (int k = 0; k < PROBDIM; k++)
            x[i * PROBDIM + k] = read_nextnum(fpin);
#if defined(SURROGATES)
		y[i] = read_nextnum(fpin);
#else
		y[i] = 0.0;
#endif
    }

	fclose(fpin);
	
    //FILE *fpout = fopen("output.knn.txt","w");

    double *d_xmem, *d_x, *d_dist;

    hipMalloc((void **)&d_xmem, TRAINELEMS*PROBDIM*sizeof(double));
    hipMalloc((void **)&d_x, QUERYELEMS*PROBDIM*sizeof(double *));
    hipMalloc((void **)&d_dist, TRAINELEMS*sizeof(double));

    hipMemcpy(d_xmem, xmem, TRAINELEMS*PROBDIM*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, QUERYELEMS*PROBDIM*sizeof(double), hipMemcpyHostToDevice);


	double *dist;
	hipHostAlloc((void **)&dist, TRAINELEMS*sizeof(double), hipHostMallocDefault);

	printf("Computing KNN...\n");
    

	double t0, t1, t_first = 0.0, t_sum = 0.0;
	double sse = 0.0;
	double err, err_sum = 0.0;

    for (int i=0;i<QUERYELEMS;i++) {	/* requests */

        t0 = gettime();
        double yp = find_knn_value(d_xmem, &d_x[i*PROBDIM], d_dist, &x[i*PROBDIM], PROBDIM, NNBS, dist);
        t1 = gettime();
        t_sum += (t1-t0);
        if (i == 0) t_first = (t1-t0);

        sse += (y[i]-yp)*(y[i]-yp);

        //for (k = 0; k < PROBDIM; k++)
        //	fprintf(fpout,"%.5f ", x[k]);

        err = 100.0*fabs((yp-y[i])/y[i]);
        //fprintf(fpout,"%.5f %.5f %.2f\n", y[i], yp, err);
        err_sum += err;
	}

	//fclose(fpout);

	double mse = sse/QUERYELEMS;
	double ymean = compute_mean(y, QUERYELEMS);
	double var = compute_var(y, QUERYELEMS, ymean);
	double r2 = 1-(mse/var);

	printf("Results for %d query points\n", QUERYELEMS);
	printf("APE = %.2f %%\n", err_sum/QUERYELEMS);
	printf("MSE = %.6f\n", mse);
	printf("R2 = 1 - (MSE/Var) = %.6lf\n", r2);

	t_sum = t_sum*1000.0;			// convert to ms
	t_first = t_first*1000.0;	// convert to ms
	printf("Total time = %lf ms\n", t_sum);
	printf("Time for 1st query = %lf ms\n", t_first);
	printf("Time for 2..N queries = %lf ms\n", t_sum-t_first);
	printf("Average time/query = %lf ms\n", (t_sum-t_first)/(QUERYELEMS-1));


	//Free the allocated memory
	// free(xmem);
	// free(xdata);
	// free(ydata);
	// free(x);
	// free(y);

	// hipFree(d_xmem);
	// hipFree(d_x);
	// hipFree(d_dist);

	return 0;
}
